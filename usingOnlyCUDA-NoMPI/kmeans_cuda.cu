#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

// CUDA kernel to assign points to the nearest cluster
__global__ void assignPointsToClusters(const double* pointsX, const double* pointsY, int* clusterAssignments,
                                        const double* centroidsX, const double* centroidsY, int numPoints, int numClusters) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    double minDist = INFINITY;
    int bestCluster = -1;

    for (int j = 0; j < numClusters; ++j) {
        double dist = (pointsX[idx] - centroidsX[j]) * (pointsX[idx] - centroidsX[j]) +
                      (pointsY[idx] - centroidsY[j]) * (pointsY[idx] - centroidsY[j]);
        if (dist < minDist) {
            minDist = dist;
            bestCluster = j;
        }
    }

    clusterAssignments[idx] = bestCluster;
}

void readInputData(vector<double>& x, vector<double>& y) {
    ifstream inputFile("input10K.txt");
    if (!inputFile.is_open()) {
        cerr << "Error opening input10K.txt file!" << endl;
        exit(EXIT_FAILURE);
    }

    double xi, yi;
    while (inputFile >> xi >> yi) {
        x.push_back(xi);
        y.push_back(yi);
    }
    inputFile.close();
}

int main() {
    // Input data variables
    vector<double> x, y;
    readInputData(x, y);
    int numPoints = x.size();

    auto start = std::chrono::high_resolution_clock::now();
    int numClusters;
    cout << "Enter the number of clusters: ";
    cin >> numClusters;
    if (numClusters <= 0 || numClusters > numPoints) {
        cerr << "Invalid number of clusters." << endl;
        return EXIT_FAILURE;
    }

    // Allocate memory for points and centroids
    double *d_pointsX, *d_pointsY;
    int *d_clusterAssignments;

    hipMalloc(&d_pointsX, numPoints * sizeof(double));
    hipMalloc(&d_pointsY, numPoints * sizeof(double));
    hipMalloc(&d_clusterAssignments, numPoints * sizeof(int));

    hipMemcpy(d_pointsX, x.data(), numPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsY, y.data(), numPoints * sizeof(double), hipMemcpyHostToDevice);

    // Initialize centroids
    vector<double> centroidsX(numClusters), centroidsY(numClusters);
    for (int i = 0; i < numClusters; ++i) {
        centroidsX[i] = x[i];
        centroidsY[i] = y[i];
    }

    double *d_centroidsX, *d_centroidsY;
    hipMalloc(&d_centroidsX, numClusters * sizeof(double));
    hipMalloc(&d_centroidsY, numClusters * sizeof(double));

    hipMemcpy(d_centroidsX, centroidsX.data(), numClusters * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroidsY, centroidsY.data(), numClusters * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (numPoints + blockSize - 1) / blockSize;

    int maxIterations = 100;
    for (int iter = 0; iter < maxIterations; ++iter) {
        // Assign points to clusters
        assignPointsToClusters<<<gridSize, blockSize>>>(d_pointsX, d_pointsY, d_clusterAssignments,
                                                        d_centroidsX, d_centroidsY, numPoints, numClusters);

        // Copy cluster assignments back to host
        vector<int> clusterAssignments(numPoints);
        hipMemcpy(clusterAssignments.data(), d_clusterAssignments, numPoints * sizeof(int), hipMemcpyDeviceToHost);

        // Compute new centroids on host
        vector<double> clusterSumsX(numClusters, 0.0), clusterSumsY(numClusters, 0.0);
        vector<int> clusterCounts(numClusters, 0);

        for (int i = 0; i < numPoints; ++i) {
            int cluster = clusterAssignments[i];
            clusterSumsX[cluster] += x[i];
            clusterSumsY[cluster] += y[i];
            clusterCounts[cluster]++;
        }

        for (int i = 0; i < numClusters; ++i) {
            if (clusterCounts[i] > 0) {
                centroidsX[i] = clusterSumsX[i] / clusterCounts[i];
                centroidsY[i] = clusterSumsY[i] / clusterCounts[i];
            }
        }

        // Copy updated centroids to device
        hipMemcpy(d_centroidsX, centroidsX.data(), numClusters * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_centroidsY, centroidsY.data(), numClusters * sizeof(double), hipMemcpyHostToDevice);
    }

    // Output final centroids
    cout << "Final Centroids after " << maxIterations << " iterations:" << endl;
    for (int i = 0; i < numClusters; ++i) {
        cout << "Cluster " << i << ": (" << centroidsX[i] << ", " << centroidsY[i] << ")" << endl;
    }

    // Free device memory
    hipFree(d_pointsX);
    hipFree(d_pointsY);
    hipFree(d_clusterAssignments);
    hipFree(d_centroidsX);
    hipFree(d_centroidsY);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;
    std::cout << "Serial Run (without MPI and CUDA) took " << duration.count() << " milliseconds." << std::endl;

    return 0;
}
